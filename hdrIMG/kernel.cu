﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <string>
#include <filesystem>

#include <fstream>
#include <iostream>
#include <string>
#include <array>
#include <vector>
#include <iterator>

using namespace std;
//using fs = std::filesystem;

__global__
void ImageFilter(int* w, int* h, int* out, int n)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n)
	{
		out[i] = w[i] + h[i];
	}
}
void ReadImages();

unsigned char* readBMP_v1(char* filename)
{
	int i;
	FILE* f = fopen(filename, "rb");
	unsigned char info[54];
	fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header

	// extract image height and width from header
	int width, height;
	memcpy(&width, info + 18, sizeof(int));
	memcpy(&height, info + 22, sizeof(int));

	int heightSign = 1;
	if (height < 0) {
		heightSign = -1;
	}

	int size = 3 * width * abs(height);
	unsigned char* data = new unsigned char[size]; // allocate 3 bytes per pixel
	fread(data, sizeof(unsigned char), size, f); // read the rest of the data at once
	fclose(f);

	if (heightSign == 1) {
		for (i = 0; i < size; i += 3)
		{
			//code to flip the image data here....
		}
	}
	return data;
}
std::vector<char> readBMP(const std::string& file)
{
	static constexpr size_t HEADER_SIZE = 54;

	std::ifstream bmp(file, std::ios::binary);

	std::array<char, HEADER_SIZE> header;
	bmp.read(header.data(), header.size());
	if (header[0] != 'B' || header[1] != 'M')
	{
		std::cout << "File is not a BMP image file" << std::endl;
		exit(0);
	}
	else
	{
		std::cout << "It's a BMP file." << endl;
	}
	auto fileSize = *reinterpret_cast<uint32_t*>(&header[2]);
	auto dataOffset = *reinterpret_cast<uint32_t*>(&header[10]);
	auto width = *reinterpret_cast<uint32_t*>(&header[18]);
	auto height = *reinterpret_cast<uint32_t*>(&header[22]);
	auto planes = *reinterpret_cast<uint16_t*>(&header[26]);
	auto depth = *reinterpret_cast<uint16_t*>(&header[28]);
	auto compression = *reinterpret_cast<uint32_t*>(&header[30]);
	auto size = *reinterpret_cast<uint32_t*>(&header[34]);
	auto h_resolution = *reinterpret_cast<uint32_t*>(&header[38]);
	auto v_resolution = *reinterpret_cast<uint32_t*>(&header[42]);
	auto num_colors = *reinterpret_cast<uint32_t*>(&header[46]);

	std::cout << "fileSize: " << fileSize << std::endl;
	std::cout << "dataOffset: " << dataOffset << std::endl;
	std::cout << "width: " << width << std::endl;
	std::cout << "height: " << height << std::endl;
	std::cout << "depth: " << depth << "-bit" << std::endl;
	std::cout << "Compresion: " << compression << std::endl;
	std::cout << "NO of planes: " << planes << std::endl;
	std::cout << "Size: " << size << std::endl;
	std::cout << "Num of colors: " << num_colors << std::endl;
	std::cout << "Horizontal resolution: " << h_resolution << std::endl;
	std::cout << "Vertical resolution: " << v_resolution << std::endl;

	// Get the current file pointer location
	std::cout << "File pointer = " << bmp.tellg() << endl;
	// Seek to data offset position
	bmp.seekg(static_cast<std::basic_istream<char, std::char_traits<char>>::off_type>(int(dataOffset)) + 20, bmp.beg);

	std::cout << "File pointer = " << bmp.tellg() << endl;


	std::vector<char> img(size);
	bmp.read(img.data(), img.size());

	char temp = 0;
	const int w = width;
	const int h = height;
	//int mat[w][h]

	for (int i = 0; i <= 10; i++)
	{
		std::cout << int(img[i] & 0xff) << endl;
	}
	//img = 

	/*
	std::vector<char> img(dataOffset - HEADER_SIZE);
	bmp.read(img.data(), img.size());

	auto dataSize = ((width * 3 + 3) & (~3)) * height;
	cout << "DataSize= " << dataSize;
	img.resize(dataSize);
	bmp.read(img.data(), img.size());

	
	char temp = 0;

	for (auto i = dataSize - 4; i >= 0; i -= 3)
	{
		temp = img[i];
		img[i] = img[i + 2];
		img[i + 2] = temp;

		std::cout << "R: " << int(img[i] & 0xff) << " G: " << int(img[i + 1] & 0xff) << " B: " << int(img[i + 2] & 0xff) << std::endl;
	}*/
	bmp.close();
	return img;
}




int main(void)
{
	int* w;
	int size = 4000;
	hipMallocManaged(&w, size);
	hipMallocManaged(&w, size);
	hipMallocManaged(&w, size);

	// call cuda kernel
	// ImageFilter <<< 1, 1>>> (w,h, out, n)
	
	// Sync
	//cudaDeviceSynchronize();

	//Free memory
	//cudaFree(w);
	ReadImages();
	//cout<< "Cplusplus: " << __cplusplus;
}

void ReadImages()
{
	string default_path = __FILE__;
	string img;
	img = "C:\\Users\\Sanjib\\Documents\\CudaProgramming\\cuda-programming\\cuda-image\\images\\image2.bmp";
	readBMP(img);
	

	struct BMap
	{
		int width;
		int height;
		int bitDepth;
	}bmp1;

}




